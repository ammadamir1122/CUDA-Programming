#include "hip/hip_runtime.h"
/* Name: task5.cu
*/
#include <stdio.h>
__global__ void myHelloOnGPU(int *array) {
array[blockIdx.x] = blockIdx.x
}
int main() {
int N = 16;
int *cpuArray = (int*)malloc(sizeof(int)*N);
int *gpuArray;
hipMalloc((void **)&gpuArray, sizeof(int)*N);
myHelloOnGPU<<<N, 1>>>(gpuArray);
hipMemcpy(cpuArray, gpuArray, sizeof(int)*N,
hipMemcpyDeviceToHost);
int i;
for (i = 0; i < N; i++) {
printf("%d ", cpuArray[i]);
}
printf("\n");
return 0;
}