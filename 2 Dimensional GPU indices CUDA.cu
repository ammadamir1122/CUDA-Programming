

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void myHelloOnGPU(int *array) {
    array[blockIdx.x] = blockIdx.x;
}
int main() {
    int N = 16;
    int *cpuArray = (int*)malloc(sizeof(int)*N);
    int *gpuArray;
    hipMalloc((void **)&gpuArray, sizeof(int)*N);
    dim3 dimGrid(N, 1, 1); dim3 dimBlock(1, 1, 1);
    myHelloOnGPU<<<dimGrid, dimBlock>>>(gpuArray);
    hipMemcpy(cpuArray, gpuArray, sizeof(int)*N,
    hipMemcpyDeviceToHost);
    int i, j;
    for (j = 0; j < N/4; j++) {
        for (i = 0; i < N/4; i++) {
            printf("%2.2d ", cpuArray[j*N/4+i]);
        }
    printf("\n");
    }
printf("\n");
return 0;
}